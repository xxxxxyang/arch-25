#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#define smalloc(type,ptr,num) if(!(ptr=(type *)malloc(sizeof(type)*(num)))) exit(1)
#define Blocksize (32)
#define Matsize (4096)
#define Verifysize (1024)
#define T (2*64)
#define U (2*64)
#define S (T/U)

__global__ void Matmul1(float *A,float *B,float *C,unsigned N){
    unsigned row=blockIdx.y*blockDim.y+threadIdx.y;
    unsigned col=blockIdx.x*blockDim.x+threadIdx.x;
    unsigned k;
    float sum=0;
    for(k=0;k<N;k++){
        sum+=A[row*N+k]*B[k*N+col];
    }
    C[row*N+col]=sum;
}

__global__ void Matmul2(float *A,float *B,float *C,unsigned N){// A,B with padding
    unsigned tx=threadIdx.x,ty=threadIdx.y;
    unsigned bx=blockIdx.x,by=blockIdx.y;
    unsigned row=by*blockDim.y+ty;
    unsigned col=bx*blockDim.x+tx;
    __shared__ float Asub[Blocksize][Blocksize],Bsub[Blocksize][Blocksize];
    float sum=0;   
    unsigned kk,k;
    for(kk=0;kk<N;kk+=Blocksize){
        Asub[ty][tx]=A[row*N+(kk+tx)];
        Bsub[ty][tx]=B[(kk+ty)*N+col];
        __syncthreads();
        for(k=0;k<Blocksize;k++){
            sum+=Asub[ty][k]*Bsub[k][tx];
        }
        __syncthreads();
    }
    C[row*N+col]=sum;
}


// 使用寄存器共享加速
__global__ void Matmul3(float *A,float *B,float *C,unsigned N){// A,B with padding
    __shared__ float B_shared[U][S];    // B block 转置共享内存
    unsigned brow=blockIdx.y, bcol=blockIdx.x;
    unsigned tid=threadIdx.y;
    // 坐标
    unsigned row=brow*T + tid;
    unsigned basecol=bcol*U;
    // 寄存器分配
    float A_reg[S] = {0}, C_reg[U] = {0};
    unsigned kk, u, s;
    for (kk = 0; kk < N; kk += S) {
        // 将 B 读入共享内存
        s = tid / U;  // 行索引
        u = tid % U;  // 列索引
        B_shared[u][s] = B[(kk + s) * N + basecol + u];
        __syncthreads();  // 所有线程等共享内存加载完
        // 将 A 读入寄存器
        #pragma unroll
        for (s = 0; s < S; s++) {
            A_reg[s] = A[row * N + kk + s];
        }
        // 计算 C 的值
        #pragma unroll
        for (u = 0; u < U; u++) {
            #pragma unroll
            for (s = 0; s < S; s++) {
                C_reg[u] += A_reg[s] * B_shared[u][s];
            }
        }
        __syncthreads();
    }
    // 写回 C
    #pragma unroll
    for (u = 0; u < U; u++) {
        unsigned global_col = basecol + u;
        C[row * N + global_col] = C_reg[u];
    }
}


__host__ void matmubase(float *A,float *B,float *C,unsigned N){
    unsigned i,j,k;
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            C[i*N+j]=0;
            for(k=0;k<N;k++){
                C[i*N+j]+=A[i*N+k]*B[k*N+j];
            }
        }
    }
}

__host__ void gen_mat(float **pA,float **pB,unsigned N){
    float *A,*B;
    smalloc(float,A,N*N);
    smalloc(float,B,N*N);
    unsigned i;
    for (i = 0; i < N*N; i++){
        A[i] = 1.0*rand()/RAND_MAX;
        B[i] = 1.0*rand()/RAND_MAX;
    }
    *pA=A;*pB=B;
}

__host__ unsigned compare(float *pred_,float *true_, unsigned n){
    unsigned i;
    float relative_error;
    for(i=0;i<n;i++){
        relative_error=fabs((pred_[i]-true_[i])/true_[i]);
        if(relative_error>=1e-6){
            printf("not equal! relative error: %12.9lf pred: %12.9f true: %12.9f\n",
                relative_error,pred_[i],true_[i]);
            return 1;
        }
    }
    printf("equal!\n");
    return 0;
}


int main(void){
    const unsigned PN=Matsize,VN=Verifysize;
    float *hA,*hB,*hC1,*hC2,*dA,*dB,*dC1,*dC2,*Cbase;
    float *hc3, *dC3;
    gen_mat(&hA,&hB,VN);
    smalloc(float,Cbase,sizeof(float)*VN*VN);
    smalloc(float,hC1,sizeof(float)*VN*VN);
    smalloc(float,hC2,sizeof(float)*VN*VN);
    smalloc(float,hc3,sizeof(float)*VN*VN);
    hipMalloc(&dA,sizeof(float)*VN*VN);
    hipMalloc(&dB,sizeof(float)*VN*VN);
    hipMalloc(&dC1,sizeof(float)*VN*VN);
    hipMalloc(&dC2,sizeof(float)*VN*VN);
    hipMalloc(&dC3,sizeof(float)*VN*VN);
    hipMemcpy(dA, hA, sizeof(float)*VN*VN, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float)*VN*VN, hipMemcpyHostToDevice);

    dim3 gridsize(VN/Blocksize,VN/Blocksize),blocksize(Blocksize,Blocksize);
    dim3 gridsize3(VN/U, VN/T), blocksize3(1, T);  // T 个线程，每个处理一行的 U 个元素
    Matmul1 <<<gridsize ,blocksize >>>(dA,dB,dC1,VN);
    Matmul2 <<<gridsize ,blocksize >>>(dA,dB,dC2,VN);
    Matmul3 <<<gridsize3,blocksize3>>>(dA,dB,dC3,VN);
    hipMemcpy(hC1, dC1, sizeof(float)*VN*VN, hipMemcpyDeviceToHost);
    hipMemcpy(hC2, dC2, sizeof(float)*VN*VN, hipMemcpyDeviceToHost);
    hipMemcpy(hc3, dC3, sizeof(float)*VN*VN, hipMemcpyDeviceToHost);
    matmubase(hA,hB,Cbase,VN);
    hipDeviceSynchronize();


    int flag=0;
    flag|=compare(hC1,Cbase,VN*VN);
    flag|=compare(hC2,Cbase,VN*VN);
    flag|=compare(hc3,Cbase,VN*VN);
    if(flag){
        printf("error!\n");
        exit(1);
    }
    printf("pass!\n");
    free(hA);free(hB);free(hC1);free(hC2);free(Cbase);free(hc3);
    hipFree(dA);hipFree(dB);hipFree(dC1);hipFree(dC2);hipFree(dC3);


    gen_mat(&hA,&hB,PN);
    hipMalloc(&dA,sizeof(float)*PN*PN);
    hipMalloc(&dB,sizeof(float)*PN*PN);
    hipMalloc(&dC1,sizeof(float)*PN*PN);
    hipMalloc(&dC2,sizeof(float)*PN*PN);
    hipMalloc(&dC3,sizeof(float)*PN*PN);
    hipMemcpy(dA, hA, sizeof(float)*PN*PN, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float)*PN*PN, hipMemcpyHostToDevice);

    gridsize={PN/Blocksize,PN/Blocksize};blocksize={Blocksize,Blocksize};
    gridsize3={PN/U,PN/T};blocksize3={1,T};

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float Time1 = 0.0,Time2=0.0,Time3=0.0,temp=0;
    const unsigned loopnum=10;
    unsigned i;
    for(i=0;i<loopnum;i++){

        hipEventRecord(start, 0);
        Matmul1 <<<gridsize,blocksize>>>(dA,dB,dC1,PN);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&temp, start, stop);
        Time1+=temp;temp=0;

        hipDeviceSynchronize();
        

        hipEventRecord(start, 0);
        Matmul2 <<<gridsize,blocksize>>>(dA,dB,dC2,PN);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&temp, start, stop);
        Time2+=temp;temp=0;

        hipDeviceSynchronize();
        
        hipEventRecord(start, 0);
        Matmul3 <<<gridsize3,blocksize3>>>(dA,dB,dC3,PN);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&temp, start, stop);
        Time3+=temp;temp=0;

        hipDeviceSynchronize();
    }
    
    printf("N: %5.d  time1: %12.9f  time2: %12.9f time3: %12.9f\n"
            ,PN,Time1/loopnum,Time2/loopnum,Time3/loopnum);
    free(hA);free(hB);
    hipFree(dA);hipFree(dB);hipFree(dC1);hipFree(dC2);hipFree(dC3);

}